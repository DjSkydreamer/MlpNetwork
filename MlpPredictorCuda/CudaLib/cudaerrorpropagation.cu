#include "hip/hip_runtime.h"
#include "cudaerrorpropagation.h"

#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"

#define A 1.2f
#define B 0.5f
#define MIN_LEARNING_RATE 0.000001f
#define MAX_LEARNING_RATE 50.0f

// Device functions

// Array[height * width] 
__device__ long index2D(int i, int j, int width)
{
	return i * width + j;
}

// Array[depth * height * width]
__device__ long index3D(int i, int j, int k, int height, int width)
{
	return (i * height + j) * width + k;
}

__device__ float unipolarSigmoidFunction(float x)
{
	return 1.0f / (1.0f + expf(-x));
}

__device__ float unipolarSigmoidDerivative(float fX)
{
	return fX * (1.0f - fX);
}

__device__ float bipolarSigmoidFunction(float x)
{
	return tanhf(x);
}

__device__ float bipolarSigmoidDerivative(float fX)
{
	return 1.0f - fX * fX;;
}

__device__ float sinusoidFunction(float x)
{
	return sinf(x);
}

__device__ float sinusoidDerivative(float fX)
{
	return sqrtf(1.0f - fX * fX);
}

__device__ float linearFunction(float x)
{
	return x;
}

__device__ float linearDerivative(float fX)
{
	return 1.0f;
}

__device__ int sign(float x)
{
	if (x > 0) return 1;
	if (x < 0) return -1;
	return 0;
}

// Pointers to device functions

__device__ func_ptr pUnipolarSigmoidFunction = unipolarSigmoidFunction;
__device__ func_ptr pUnipolarSigmoidDerivative = unipolarSigmoidDerivative;

__device__ func_ptr pBipolarSigmoidFunction = bipolarSigmoidFunction;
__device__ func_ptr pBipolarSigmoidDerivative = bipolarSigmoidDerivative;

__device__ func_ptr pSinusoidFunction = sinusoidFunction;
__device__ func_ptr pSinusoidDerivative = sinusoidDerivative;

__device__ func_ptr pLinearFunction = linearFunction;
__device__ func_ptr pLinearDerivative = linearDerivative;

// Cuda kernels

__global__ void computeLayerOutputBatchKernel(func_ptr layerActivationFunc, const float *layerInsBatch /*2d*/,
	const float *layerWeights /*2d*/, float *layerOutsBatch /*2d*/, int numLayerInput, int numLayerOutput, int numSamples)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= numLayerOutput || k >= numSamples)
		return;

	float sum = layerWeights[index2D(0, j, numLayerOutput)] * 1.0f; // bias
	for (int i = 0; i < numLayerInput; ++i)
	{
		sum += layerWeights[index2D((i + 1), j, numLayerOutput)] * layerInsBatch[index2D(k, i, numLayerInput)];
	}

	layerOutsBatch[index2D(k, j, numLayerOutput)] = layerActivationFunc(sum);
}

__global__ void computeHOGradsBatchKernel(func_ptr outputFuncDerivative, float *hoGradsBatch /*3d*/, float *errorsOutsBatch /*2d*/,
	float *oDeltasBatch /*2d*/, const float *hOutsBatch /*2d*/, const float *netOutsBatch /*2d*/, const float *targetOutsBatch /*2d*/,
	int numHidden, int numOutput, int numSamples)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= numOutput || k >= numSamples)
		return;

	float error = (netOutsBatch[index2D(k, j, numOutput)] - targetOutsBatch[index2D(k, j, numOutput)]);

	errorsOutsBatch[index2D(k, j, numOutput)] = error * error;

	oDeltasBatch[index2D(k, j, numOutput)] = error * outputFuncDerivative(netOutsBatch[index2D(k, j, numOutput)]);

	hoGradsBatch[index3D(k, 0, j, (numHidden + 1), numOutput)] = oDeltasBatch[index2D(k, j, numOutput)] * 1.0f; // bias
	for (int i = 0; i < numHidden; ++i)
	{
		hoGradsBatch[index3D(k, (i + 1), j, (numHidden + 1), numOutput)] = oDeltasBatch[index2D(k, j, numOutput)] * hOutsBatch[index2D(k, i, numHidden)];
	}
}

__global__ void computeIHGradsBatchKernel(func_ptr hiddenFuncDerivative, float *ihGradsBatch /*3d*/,
	const float *errorsOutsBatch /*2d*/, float *errorsBatch, const float *hoWeights /*2d*/, const float *oDeltasBatch /*2d*/,
	float *hDeltasBatch /*2d*/, const float *hOutsBatch /*2d*/, const float *netInsBatch /*2d*/,
	int numInput, int numHidden, int numOutput, int numSamples)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numInput + 1 /* bias */) || k >= numSamples)
		return;

	float input = ((k % numInput == 0) && (i == 0)) ? 1.0f : netInsBatch[index2D(k, i - 1, numInput)]; // bias?
	for (int j = 0; j < numHidden; ++j)
	{
		float sum = 0.0f;
		float error = 0.0f;
		for (int s = 0; s < numOutput; ++s)
		{
			sum += oDeltasBatch[index2D(k, s, numOutput)] * hoWeights[index2D((j + 1), s, numOutput)];
			error += errorsOutsBatch[index2D(k, s, numOutput)];
		}
		errorsBatch[k] = error;

		hDeltasBatch[index2D(k, j, numHidden)] = sum * hiddenFuncDerivative(hOutsBatch[index2D(k, j, numHidden)]);
		ihGradsBatch[index3D(k, i, j, (numInput + 1), numHidden)] = hDeltasBatch[index2D(k, j, numHidden)] * input;
	}
}

__global__ void computeLayerGradsKernel(float *layerGrads /*2d*/, float *layerGradsBatch /*3d*/,
	float *error /* Single value */, float *errorsBatch, float *layerWeights /*2d*/, int numLayerInput, int numLayerOutput,
	int numSamples, bool computeError)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numLayerInput + 1 /* bias */) || j >= numLayerOutput)
		return;

	bool computeErrorOnFirstIteration = (computeError && i == 0 && j == 0) ? true : false;

	float gradsSum = 0.0f;
	if (computeErrorOnFirstIteration)
		*error = 0.0f;
	for (int k = 0; k < numSamples; ++k)
	{
		gradsSum += layerGradsBatch[index3D(k, i, j, (numLayerInput + 1), numLayerOutput)];

		if (computeErrorOnFirstIteration)
			*error += errorsBatch[k];
	}

	layerGrads[index2D(i, j, numLayerOutput)] = gradsSum;
}

__global__ void updateLayerWeightsBackPropKernel(const float *layerGrads /*2d*/, float *layerWeights /*2d*/,
	float *prevLayerWeightDeltas /*2d*/, float learningRate, float momentum, int numLayerInput, int numLayerOutput)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numLayerInput + 1 /* bias */) || j >= numLayerOutput)
		return;

	float deltaW = -learningRate * layerGrads[index2D(i, j, numLayerOutput)];
	layerWeights[index2D(i, j, numLayerOutput)] += deltaW;
	layerWeights[index2D(i, j, numLayerOutput)] += momentum * prevLayerWeightDeltas[index2D(i, j, numLayerOutput)];
	prevLayerWeightDeltas[index2D(i, j, numLayerOutput)] = deltaW;
}

__global__ void updateLayerWeightsResilientPropKernel(const float *layerGrads /*2d*/, const float *prevLayerGrads /*2d*/,
	float *layerWeights /*2d*/, float *layerLearningRates /*2d*/, int numLayerInput, int numLayerOutput)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numLayerInput + 1 /* bias */) || j >= numLayerOutput)
		return;

	float previousGradient = prevLayerGrads[index2D(i, j, numLayerOutput)];
	float currentGradient = layerGrads[index2D(i, j, numLayerOutput)];
	float change = previousGradient * currentGradient;

	if (change > 0)
	{
		layerLearningRates[index2D(i, j, numLayerOutput)] = fminf(A * layerLearningRates[index2D(i, j, numLayerOutput)], MAX_LEARNING_RATE);
	}
	else if (change < 0)
	{
		layerLearningRates[index2D(i, j, numLayerOutput)] = fmaxf(B * layerLearningRates[index2D(i, j, numLayerOutput)], MIN_LEARNING_RATE);
	}

	float deltaW = -layerLearningRates[index2D(i, j, numLayerOutput)] * sign(currentGradient);
	layerWeights[index2D(i, j, numLayerOutput)] += deltaW;
}

// Make randomly generated weights in (0.0, 1.0] be in the interval from -maxAbs to +maxAbs.
__global__ void normalizeLayerWeightsKernel(float *layerWeights /*2d*/, float maxAbs, int numLayerWeights)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= numLayerWeights)
		return;

	layerWeights[i] = ((layerWeights[i] - 0.5f) / 0.5f) * maxAbs;
}

int computeNumBlocks(int dataSize, int threadsPerBlock)
{
	int numBlocks = dataSize / threadsPerBlock;

	if (dataSize % threadsPerBlock)
		numBlocks++;

	return numBlocks;

	//return (dataSize + threadsPerBlock - 1) / threadsPerBlock;
}

dim3 getBlockDim1D()
{
	return dim3(16);
}

dim3 getBlockDim2D()
{
	return dim3(16, 16);
}

dim3 getGridDim1D(int dataSizeX, int threadsPerBlockX)
{
	return dim3(computeNumBlocks(dataSizeX, threadsPerBlockX));
}

dim3 getGridDim2D(int dataSizeX, int threadsPerBlockX, int dataSizeY, int threadsPerBlockY)
{
	return dim3(computeNumBlocks(dataSizeX, threadsPerBlockX), computeNumBlocks(dataSizeY, threadsPerBlockY));
}

void generateRandomFloatArrays(float *array1 /*2d*/, float *array2 /*2d*/, int array1Size, int array2Size)
{
	unsigned long long seed = (unsigned long long)time(NULL);

	hiprandGenerator_t gen;

	// Create and initialize generator
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

	hiprandGenerateUniform(gen, array1, array1Size);
	hiprandGenerateUniform(gen, array2, array2Size);

	hiprandDestroyGenerator(gen);
}

void normalizeWeights(float *d_inputHiddenWeights /*2d*/, float *d_hiddenOutputWeights /*2d*/,
	int numInputHiddenWeights, int numHiddenOutputWeights)
{
	dim3 blockDim = getBlockDim1D();

	dim3 gridDim1 = getGridDim1D(numInputHiddenWeights, blockDim.x);
	normalizeLayerWeightsKernel<<<gridDim1, blockDim>>>(d_inputHiddenWeights, 1.0f, numInputHiddenWeights);

	dim3 gridDim2 = getGridDim1D(numHiddenOutputWeights, blockDim.x);
	normalizeLayerWeightsKernel<<<gridDim2, blockDim>>>(d_hiddenOutputWeights, 1.0f, numHiddenOutputWeights);
}

void randomizeWeights(CudaErrorPropagation *propagation)
{
	float *d_inputHiddenWeights = propagation->d_inputHiddenWeights;
	float *d_hiddenOutputWeights = propagation->d_hiddenOutputWeights;
	int numInputHiddenWeights = (propagation->numInput + 1) * propagation->numHidden;
	int numHiddenOutputWeights = (propagation->numHidden + 1) * propagation->numOutput;

	generateRandomFloatArrays(d_inputHiddenWeights, d_hiddenOutputWeights, numInputHiddenWeights, numHiddenOutputWeights);
	normalizeWeights(d_inputHiddenWeights, d_hiddenOutputWeights, numInputHiddenWeights, numHiddenOutputWeights);
}

void randomizeLearningRates(CudaErrorPropagation *propagation)
{
	float *d_inputHiddenLearningRates = propagation->d_inputHiddenLearningRates;
	float *d_hiddenOutputLearningRates = propagation->d_hiddenOutputLearningRates;
	int numInputHiddenLearningRates = (propagation->numInput + 1) * propagation->numHidden;
	int numHiddenOutputLearningRates = (propagation->numHidden + 1) * propagation->numOutput;

	generateRandomFloatArrays(d_inputHiddenLearningRates, d_hiddenOutputLearningRates, numInputHiddenLearningRates,
		numHiddenOutputLearningRates);
}

void setLayerFunctionAndDerivative(func_ptr *function, func_ptr *derivative, ActivationFuncType type)
{
	switch (type)
	{
	case ActivationFuncType::UNIPOLAR_SIGMOID:
		hipMemcpyFromSymbol(function, HIP_SYMBOL(pUnipolarSigmoidFunction ), sizeof(func_ptr));
		hipMemcpyFromSymbol(derivative, HIP_SYMBOL(pUnipolarSigmoidDerivative), sizeof(func_ptr));
		break;
	case ActivationFuncType::BIPOLAR_SIGMOID:
		hipMemcpyFromSymbol(function, HIP_SYMBOL(pBipolarSigmoidFunction), sizeof(func_ptr));
		hipMemcpyFromSymbol(derivative, HIP_SYMBOL(pBipolarSigmoidDerivative), sizeof(func_ptr));
		break;
	case ActivationFuncType::SINUSOID:
		hipMemcpyFromSymbol(function, HIP_SYMBOL(pSinusoidFunction), sizeof(func_ptr));
		hipMemcpyFromSymbol(derivative, HIP_SYMBOL(pSinusoidDerivative), sizeof(func_ptr));
		break;
	case ActivationFuncType::LINEAR:
		hipMemcpyFromSymbol(function, HIP_SYMBOL(pLinearFunction), sizeof(func_ptr));
		hipMemcpyFromSymbol(derivative, HIP_SYMBOL(pLinearDerivative), sizeof(func_ptr));
		break;
	default:
		hipMemcpyFromSymbol(function, HIP_SYMBOL(pUnipolarSigmoidFunction), sizeof(func_ptr));
		hipMemcpyFromSymbol(derivative, HIP_SYMBOL(pUnipolarSigmoidDerivative), sizeof(func_ptr));
		break;
	}
}

bool checkCudaSupport()
{
	int deviceCount, device;
	int gpuDeviceCount = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device)
	{
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) /* 9999 means emulation only */
			++gpuDeviceCount;
	}

	/* don't just return the number of gpus, because other runtime cuda
	errors can also yield non-zero return values */
	if (gpuDeviceCount > 0)
		return true; /* success */
	else
		return false; /* failure */
}

CudaErrorPropagation* createErrorPropagation(float *h_inputData /*2d*/, float *h_outputData /*2d*/,
	float *h_inputHiddenWeights /*2d*/, float *h_hiddenOutputWeights /*2d*/,
	int numInput, int numHidden, int numOutput, int numSamples,
	ActivationFuncType hiddenFunc, ActivationFuncType outputFunc)
{
	CudaErrorPropagation *propagation = (CudaErrorPropagation *) malloc(sizeof(CudaErrorPropagation));

	// Initialize network and data params 
	propagation->numInput = numInput;
	propagation->numHidden = numHidden;
	propagation->numOutput = numOutput;
	propagation->numSamples = numSamples;

	hipMalloc((void**) &(propagation->d_inputsBatch), numSamples * numInput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenWeights), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputsBatch), numSamples * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputWeights), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_outputsBatch), numSamples * numOutput * sizeof(float));

	// Propagation
	hipMalloc((void**) &(propagation->d_targetOutputsBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_outputDeltasBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputGradients), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenDeltasBatch), numSamples * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenGradients), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_errorsOutputsBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_errorsBatch), numSamples * sizeof(float));
	hipMalloc((void**) &(propagation->d_error), sizeof(float));

	hipMalloc((void**) &(propagation->d_hiddenOutputGradientsBatch), numSamples * (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenGradientsBatch), numSamples * (numInput + 1) * numHidden * sizeof(float));

	// BackPropagation
	hipMalloc((void**) &(propagation->d_previousInputHiddenWeightDeltas), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_previousHiddenOutputWeightDeltas), (numHidden + 1) * numOutput * sizeof(float));

	// ResilientPropagation
	hipMalloc((void**) &(propagation->d_previousInputHiddenGradients), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_previousHiddenOutputGradients), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenLearningRates), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputLearningRates), (numHidden + 1) * numOutput * sizeof(float));

	// Computed weights
	propagation->h_inputHiddenWeights = (float *) malloc((numInput + 1) * numHidden * sizeof(float));
	propagation->h_hiddenOutputWeights = (float *) malloc((numHidden + 1) * numOutput * sizeof(float));

	// Copy initial network weights
	memcpy(propagation->h_inputHiddenWeights, h_inputHiddenWeights, (numInput + 1) * numHidden * sizeof(float));
	memcpy(propagation->h_hiddenOutputWeights, h_hiddenOutputWeights, (numHidden + 1) * numOutput * sizeof(float));
	hipMemcpy(propagation->d_inputHiddenWeights, propagation->h_inputHiddenWeights,
		(propagation->numInput + 1) * propagation->numHidden * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(propagation->d_hiddenOutputWeights, propagation->h_hiddenOutputWeights,
		(propagation->numHidden + 1) * propagation->numOutput * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);

	// Copy input and output learning data
	hipMemcpy(propagation->d_inputsBatch, h_inputData, numSamples * numInput * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(propagation->d_targetOutputsBatch, h_outputData, numSamples * numOutput * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);

	// Reset previous params to 0
	hipMemset(propagation->d_previousInputHiddenWeightDeltas, 0, (numInput + 1) * numHidden * sizeof(float));
	hipMemset(propagation->d_previousHiddenOutputWeightDeltas, 0, (numHidden + 1) * numOutput * sizeof(float));
	hipMemset(propagation->d_previousInputHiddenGradients, 0, (numInput + 1) * numHidden * sizeof(float));
	hipMemset(propagation->d_previousHiddenOutputGradients, 0, (numHidden + 1) * numOutput * sizeof(float));

	randomizeLearningRates(propagation);

	// Set layers activation functions and derivatives
	setLayerFunctionAndDerivative(&(propagation->h_pHiddenFunction), &(propagation->h_pHiddenDerivative), hiddenFunc);
	setLayerFunctionAndDerivative(&(propagation->h_pOutputFunction), &(propagation->h_pOutputDerivative), outputFunc);
	
	return propagation;
}

void destroyErrorPropagation(CudaErrorPropagation *propagation)
{
	if (!propagation)
		return;
	// Network and data
	hipFree(propagation->d_inputsBatch);
	hipFree(propagation->d_inputHiddenWeights);
	hipFree(propagation->d_hiddenOutputsBatch);
	hipFree(propagation->d_hiddenOutputWeights);
	hipFree(propagation->d_outputsBatch);

	// Propagation
	hipFree(propagation->d_targetOutputsBatch);
	hipFree(propagation->d_outputDeltasBatch);
	hipFree(propagation->d_hiddenOutputGradients);
	hipFree(propagation->d_hiddenDeltasBatch);
	hipFree(propagation->d_inputHiddenGradients);
	hipFree(propagation->d_errorsOutputsBatch);
	hipFree(propagation->d_errorsBatch);
	hipFree(propagation->d_error);

	hipFree(propagation->d_hiddenOutputGradientsBatch);
	hipFree(propagation->d_inputHiddenGradientsBatch);

	// BackPropagation
	hipFree(propagation->d_previousInputHiddenWeightDeltas);
	hipFree(propagation->d_previousHiddenOutputWeightDeltas);

	// ResilientPropagation
	hipFree(propagation->d_previousInputHiddenGradients);
	hipFree(propagation->d_previousHiddenOutputGradients);
	hipFree(propagation->d_inputHiddenLearningRates);
	hipFree(propagation->d_hiddenOutputLearningRates);

	// Computed weights
	free(propagation->h_inputHiddenWeights);
	free(propagation->h_hiddenOutputWeights);

	free(propagation);
}

const float* getInputHiddenWeights(CudaErrorPropagation *propagation)
{
	hipMemcpy(propagation->h_inputHiddenWeights, propagation->d_inputHiddenWeights,
		(propagation->numInput + 1) * propagation->numHidden * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	return propagation->h_inputHiddenWeights;
}

const float* getHiddenOutputWeights(CudaErrorPropagation *propagation)
{
	hipMemcpy(propagation->h_hiddenOutputWeights, propagation->d_hiddenOutputWeights,
		(propagation->numHidden + 1) * propagation->numOutput * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	return propagation->h_hiddenOutputWeights;
}

void computeOutputBatch(CudaErrorPropagation *propagation)
{
	dim3 blockDim = getBlockDim2D();

	dim3 gridDim1 = getGridDim2D(propagation->numHidden, blockDim.x, propagation->numSamples, blockDim.y);	
	computeLayerOutputBatchKernel<<<gridDim1, blockDim>>>(propagation->h_pHiddenFunction, propagation->d_inputsBatch,
		propagation->d_inputHiddenWeights, propagation->d_hiddenOutputsBatch,
		propagation->numInput, propagation->numHidden, propagation->numSamples);

	dim3 gridDim2 = getGridDim2D(propagation->numOutput, blockDim.x, propagation->numSamples, blockDim.y);	
	computeLayerOutputBatchKernel<<<gridDim2, blockDim>>>(propagation->h_pOutputFunction, propagation->d_hiddenOutputsBatch,
		propagation->d_hiddenOutputWeights, propagation->d_outputsBatch,
		propagation->numHidden, propagation->numOutput, propagation->numSamples);
}

void computeGradients(CudaErrorPropagation *propagation)
{
	dim3 blockDim = getBlockDim2D();

	dim3 gridDim1 = getGridDim2D(propagation->numOutput, blockDim.x, propagation->numSamples, blockDim.y);
	computeHOGradsBatchKernel<<<gridDim1, blockDim>>>(propagation->h_pOutputDerivative, propagation->d_hiddenOutputGradientsBatch,
		propagation->d_errorsOutputsBatch, propagation->d_outputDeltasBatch, propagation->d_hiddenOutputsBatch,
		propagation->d_outputsBatch, propagation->d_targetOutputsBatch, propagation->numHidden, propagation->numOutput,
		propagation->numSamples);

	dim3 gridDim2 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numSamples, blockDim.y);
	computeIHGradsBatchKernel<<<gridDim2, blockDim>>>(propagation->h_pHiddenDerivative, propagation->d_inputHiddenGradientsBatch,
		propagation->d_errorsOutputsBatch, propagation->d_errorsBatch, propagation->d_hiddenOutputWeights,
		propagation->d_outputDeltasBatch, propagation->d_hiddenDeltasBatch, propagation->d_hiddenOutputsBatch,
		propagation->d_inputsBatch, propagation->numInput, propagation->numHidden, propagation->numOutput, propagation->numSamples);

	dim3 gridDim3 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numHidden, blockDim.y);
	computeLayerGradsKernel<<<gridDim3, blockDim>>>(propagation->d_inputHiddenGradients, propagation->d_inputHiddenGradientsBatch,
		propagation->d_error, propagation->d_errorsBatch, propagation->d_inputHiddenWeights,
		propagation->numInput, propagation->numHidden, propagation->numSamples, true);

	dim3 gridDim4 = getGridDim2D(propagation->numHidden + 1 /* bias */, blockDim.x, propagation->numOutput, blockDim.y);
	computeLayerGradsKernel<<<gridDim4, blockDim>>>(propagation->d_hiddenOutputGradients, propagation->d_hiddenOutputGradientsBatch,
		propagation->d_error, propagation->d_errorsBatch, propagation->d_hiddenOutputWeights,
		propagation->numHidden, propagation->numOutput, propagation->numSamples, false);
}

void updateWeightsBackProp(CudaErrorPropagation *propagation, float learningRate, float momentum)
{
	dim3 blockDim = getBlockDim2D();
	
	dim3 gridDim1 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numHidden, blockDim.y);
	updateLayerWeightsBackPropKernel<<<gridDim1, blockDim>>>(propagation->d_inputHiddenGradients, propagation->d_inputHiddenWeights,
		propagation->d_previousInputHiddenWeightDeltas, learningRate, momentum, propagation->numInput, propagation->numHidden);

	dim3 gridDim2 = getGridDim2D(propagation->numHidden + 1 /* bias */, blockDim.x, propagation->numOutput, blockDim.y);
	updateLayerWeightsBackPropKernel<<<gridDim2, blockDim>>>(propagation->d_hiddenOutputGradients, propagation->d_hiddenOutputWeights,
		propagation->d_previousHiddenOutputWeightDeltas, learningRate, momentum, propagation->numHidden, propagation->numOutput);
}

void updateWeightsResilientProp(CudaErrorPropagation *propagation)
{
	dim3 blockDim = getBlockDim2D();

	dim3 gridDim1 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numHidden, blockDim.y);
	updateLayerWeightsResilientPropKernel<<<gridDim1, blockDim>>>(propagation->d_inputHiddenGradients,
		propagation->d_previousInputHiddenGradients, propagation->d_inputHiddenWeights, propagation->d_inputHiddenLearningRates,
		propagation->numInput, propagation->numHidden);

	dim3 gridDim2 = getGridDim2D(propagation->numHidden + 1 /* bias */, blockDim.x, propagation->numOutput, blockDim.y);
	updateLayerWeightsResilientPropKernel<<<gridDim2, blockDim>>>(propagation->d_hiddenOutputGradients,
		propagation->d_previousHiddenOutputGradients, propagation->d_hiddenOutputWeights, propagation->d_hiddenOutputLearningRates,
		propagation->numHidden, propagation->numOutput);
}

float performBackPropEpoch(CudaErrorPropagation *propagation, float learningRate, float momentum)
{
	computeOutputBatch(propagation);
	computeGradients(propagation);
	updateWeightsBackProp(propagation, learningRate, momentum);

	float h_error = FLT_MAX;
	hipError_t status = hipMemcpy(&h_error, propagation->d_error, sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	if (status != hipError_t::hipSuccess)
		return 1.0f;

	//return h_error * 0.5f;
	return sqrtf((1.0f / propagation->numSamples) * (1.0f / propagation->numOutput) * h_error);
}

float performResilientPropEpoch(CudaErrorPropagation *propagation)
{
	computeOutputBatch(propagation);
	computeGradients(propagation);
	updateWeightsResilientProp(propagation);

	float h_error = FLT_MAX;
	hipError_t status = hipMemcpy(&h_error, propagation->d_error, sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	if (status != hipError_t::hipSuccess)
		return 1.0f;

	//return h_error * 0.5f;
	return sqrtf((1.0f / propagation->numSamples) * (1.0f / propagation->numOutput) * h_error);
}