#include "hip/hip_runtime.h"
#include "cudaerrorpropagation.h"

#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"

#define A 1.2f
#define B 0.5f
#define MIN_LEARNING_RATE 0.000001f
#define MAX_LEARNING_RATE 50.0f

// Array[height * width] 
__device__ long index2D(int i, int j, int width)
{
	return i * width + j;
}

// Array[depth * height * width]
__device__ long index3D(int i, int j, int k, int height, int width)
{
	return (i * height + j) * width + k;
}

__device__ float unipolarSigmoidFunction(float x)
{
	return 1.0f / (1.0f + expf(-x));
}

__device__ float unipolarSigmoidDerivative(float fX)
{
	return fX * (1.0f - fX);
}

__device__ int sign(float x)
{
	if (x > 0) return 1;
	if (x < 0) return -1;
	return 0;
}

__global__ void computeLayerOutputBatchKernel(const float *layerInsBatch /*2d*/,
	const float *layerWeights /*2d*/, float *layerOutsBatch /*2d*/, int numLayerInput, int numLayerOutput, int numSamples)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= numLayerOutput || k >= numSamples)
		return;

	float sum = layerWeights[index2D(0, j, numLayerOutput)] * 1.0f; // bias
	for (int i = 0; i < numLayerInput; ++i)
	{
		sum += layerWeights[index2D((i + 1), j, numLayerOutput)] * layerInsBatch[index2D(k, i, numLayerInput)];
	}

	layerOutsBatch[index2D(k, j, numLayerOutput)] = unipolarSigmoidFunction(sum);
}

__global__ void computeHOGradsBatchKernel(float *hoGradsBatch /*3d*/, float *errorsOutsBatch /*2d*/, float *oDeltasBatch /*2d*/,
	const float *hOutsBatch /*2d*/, const float *netOutsBatch /*2d*/, const float *targetOutsBatch /*2d*/,
	int numHidden, int numOutput, int numSamples)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= numOutput || k >= numSamples)
		return;

	float error = (netOutsBatch[index2D(k, j, numOutput)] - targetOutsBatch[index2D(k, j, numOutput)]);

	errorsOutsBatch[index2D(k, j, numOutput)] = error * error;

	oDeltasBatch[index2D(k, j, numOutput)] = error * unipolarSigmoidDerivative(netOutsBatch[index2D(k, j, numOutput)]);

	hoGradsBatch[index3D(k, 0, j, (numHidden + 1), numOutput)] = oDeltasBatch[index2D(k, j, numOutput)] * 1.0f; // bias
	for (int i = 0; i < numHidden; ++i)
	{
		hoGradsBatch[index3D(k, (i + 1), j, (numHidden + 1), numOutput)] = oDeltasBatch[index2D(k, j, numOutput)] * hOutsBatch[index2D(k, i, numHidden)];
	}
}

__global__ void computeIHGradsBatchKernel(float *ihGradsBatch /*3d*/, const float *errorsOutsBatch /*2d*/,
	float *errorsBatch, const float *hoWeights /*2d*/, const float *oDeltasBatch /*2d*/, float *hDeltasBatch /*2d*/,
	const float *hOutsBatch /*2d*/, const float *netInsBatch /*2d*/, int numInput, int numHidden, int numOutput, int numSamples)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numInput + 1 /* bias */) || k >= numSamples)
		return;

	float input = ((k % numInput == 0) && (i == 0)) ? 1.0f : netInsBatch[index2D(k, i - 1, numInput)]; // bias?
	for (int j = 0; j < numHidden; ++j)
	{
		float sum = 0.0f;
		float error = 0.0f;
		for (int s = 0; s < numOutput; ++s)
		{
			sum += oDeltasBatch[index2D(k, s, numOutput)] * hoWeights[index2D((j + 1), s, numOutput)];
			error += errorsOutsBatch[index2D(k, s, numOutput)];
		}
		errorsBatch[k] = error;

		hDeltasBatch[index2D(k, j, numHidden)] = sum * unipolarSigmoidDerivative(hOutsBatch[index2D(k, j, numHidden)]);
		ihGradsBatch[index3D(k, i, j, (numInput + 1), numHidden)] = hDeltasBatch[index2D(k, j, numHidden)] * input;
	}
}

__global__ void computeLayerGradsKernel(float *layerGrads /*2d*/, float *layerGradsBatch /*3d*/,
	float *error /* Single value */, float *errorsBatch, float *layerWeights /*2d*/, int numLayerInput, int numLayerOutput,
	int numSamples, bool computeError)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numLayerInput + 1 /* bias */) || j >= numLayerOutput)
		return;

	bool computeErrorOnFirstIteration = (computeError && i == 0 && j == 0) ? true : false;

	float gradsSum = 0.0f;
	if (computeErrorOnFirstIteration)
		*error = 0.0f;
	for (int k = 0; k < numSamples; ++k)
	{
		gradsSum += layerGradsBatch[index3D(k, i, j, (numLayerInput + 1), numLayerOutput)];

		if (computeErrorOnFirstIteration)
			*error += errorsBatch[k];
	}

	layerGrads[index2D(i, j, numLayerOutput)] = gradsSum;
}

__global__ void updateLayerWeightsBackPropKernel(const float *layerGrads /*2d*/, float *layerWeights /*2d*/,
	float *prevLayerWeightDeltas /*2d*/, float learningRate, float momentum, int numLayerInput, int numLayerOutput)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= (numLayerInput + 1 /* bias */) || j >= numLayerOutput)
		return;

	float deltaW = -learningRate * layerGrads[index2D(i, j, numLayerOutput)];
	layerWeights[index2D(i, j, numLayerOutput)] += deltaW;
	layerWeights[index2D(i, j, numLayerOutput)] += momentum * prevLayerWeightDeltas[index2D(i, j, numLayerOutput)];
	prevLayerWeightDeltas[index2D(i, j, numLayerOutput)] = deltaW;
}

//private void UpdateLayerWeights(float [] [] layerGradients, float [] [] previousLayerGradients,
//	float [] [] layerWeights, float [] [] layerLearningRates, int numLayerInput, int numLayerOutput)
//{
//	for (int i = 0; i < numLayerInput + 1; i++)
//	{
//		for (int j = 0; j < numLayerOutput; j++)
//		{
//			float previousGradient = previousLayerGradients[i][j];
//			float currentGradient = layerGradients[i][j];
//			float change = previousGradient * currentGradient;
//
//			if (change > 0)
//			{
//				layerLearningRates[i][j] = Math.Min(A * layerLearningRates[i][j], MaxLearningRate);
//			}
//			else if (change < 0)
//			{
//				layerLearningRates[i][j] = Math.Max(B * layerLearningRates[i][j], MinLearningRate);
//			}
//
//			float deltaW = -layerLearningRates[i][j] * Sign(currentGradient);
//			layerWeights[i][j] += deltaW;
//		}
//	}
//}

// Make randomly generated weights in (0.0, 1.0] be in the interval from -maxAbs to +maxAbs.
__global__ void normalizeLayerWeightsKernel(float *layerWeights /*2d*/, float maxAbs, int numLayerWeights)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= numLayerWeights)
		return;

	layerWeights[i] = ((layerWeights[i] - 0.5f) / 0.5f) * maxAbs;
}

int computeNumBlocks(int dataSize, int threadsPerBlock)
{
	int numBlocks = dataSize / threadsPerBlock;

	if (dataSize % threadsPerBlock)
		numBlocks++;

	return numBlocks;

	//return (dataSize + threadsPerBlock - 1) / threadsPerBlock;
}

dim3 getBlockDim1D()
{
	return dim3(16);
}

dim3 getBlockDim2D()
{
	return dim3(16, 16);
}

dim3 getGridDim1D(int dataSizeX, int threadsPerBlockX)
{
	return dim3(computeNumBlocks(dataSizeX, threadsPerBlockX));
}

dim3 getGridDim2D(int dataSizeX, int threadsPerBlockX, int dataSizeY, int threadsPerBlockY)
{
	return dim3(computeNumBlocks(dataSizeX, threadsPerBlockX), computeNumBlocks(dataSizeY, threadsPerBlockY));
}

void generateRandomFloatArrays(float *array1 /*2d*/, float *array2 /*2d*/, int array1Size, int array2Size)
{
	long seed = time(NULL);

	hiprandGenerator_t gen;

	// Create and initialize generator
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

	hiprandGenerateUniform(gen, array1, array1Size);
	hiprandGenerateUniform(gen, array2, array2Size);

	hiprandDestroyGenerator(gen);
}

void normalizeWeights(float *d_inputHiddenWeights /*2d*/, float *d_hiddenOutputWeights /*2d*/,
	int numInputHiddenWeights, int numHiddenOutputWeights)
{
	dim3 blockDim = getBlockDim1D();

	dim3 gridDim1 = getGridDim1D(numInputHiddenWeights, blockDim.x);
	normalizeLayerWeightsKernel<<<gridDim1, blockDim>>>(d_inputHiddenWeights, 1.0f, numInputHiddenWeights);

	dim3 gridDim2 = getGridDim1D(numHiddenOutputWeights, blockDim.x);
	normalizeLayerWeightsKernel<<<gridDim2, blockDim>>>(d_hiddenOutputWeights, 1.0f, numHiddenOutputWeights);
}

void randomizeWeights(CudaErrorPropagation *propagation)
{
	float *d_inputHiddenWeights = propagation->d_inputHiddenWeights;
	float *d_hiddenOutputWeights = propagation->d_hiddenOutputWeights;
	int numInputHiddenWeights = (propagation->numInput + 1) * propagation->numHidden;
	int numHiddenOutputWeights = (propagation->numHidden + 1) * propagation->numOutput;

	generateRandomFloatArrays(d_inputHiddenWeights, d_hiddenOutputWeights, numInputHiddenWeights, numHiddenOutputWeights);
	normalizeWeights(d_inputHiddenWeights, d_hiddenOutputWeights, numInputHiddenWeights, numHiddenOutputWeights);
}

void randomizeLearningRates(CudaErrorPropagation *propagation)
{
	float *d_inputHiddenLearningRates = propagation->d_inputHiddenLearningRates;
	float *d_hiddenOutputLearningRates = propagation->d_hiddenOutputLearningRates;
	int numInputHiddenLearningRates = (propagation->numInput + 1) * propagation->numHidden;
	int numHiddenOutputLearningRates = (propagation->numHidden + 1) * propagation->numOutput;

	generateRandomFloatArrays(d_inputHiddenLearningRates, d_hiddenOutputLearningRates, numInputHiddenLearningRates,
		numHiddenOutputLearningRates);
}

CudaErrorPropagation* createErrorPropagation(float *h_inputData /*2d*/, float *h_outputData /*2d*/,
	float *h_inputHiddenWeights /*2d*/, float *h_hiddenOutputWeights /*2d*/,
	int numInput, int numHidden, int numOutput, int numSamples)
{
	CudaErrorPropagation *propagation = (CudaErrorPropagation *) malloc(sizeof(CudaErrorPropagation));

	// Network and data
	propagation->numInput = numInput;
	propagation->numHidden = numHidden;
	propagation->numOutput = numOutput;
	propagation->numSamples = numSamples;

	hipMalloc((void**) &(propagation->d_inputsBatch), numSamples * numInput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenWeights), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputsBatch), numSamples * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputWeights), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_outputsBatch), numSamples * numOutput * sizeof(float));

	// Propagation
	hipMalloc((void**) &(propagation->d_targetOutputsBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_outputDeltasBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputGradients), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenDeltasBatch), numSamples * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenGradients), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_errorsOutputsBatch), numSamples * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_errorsBatch), numSamples * sizeof(float));
	hipMalloc((void**) &(propagation->d_error), sizeof(float));

	hipMalloc((void**) &(propagation->d_hiddenOutputGradientsBatch), numSamples * (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenGradientsBatch), numSamples * (numInput + 1) * numHidden * sizeof(float));

	// BackPropagation
	hipMalloc((void**) &(propagation->d_previousInputHiddenWeightDeltas), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_previousHiddenOutputWeightDeltas), (numHidden + 1) * numOutput * sizeof(float));

	// ResilientPropagation
	hipMalloc((void**) &(propagation->d_previousInputHiddenGradients), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_previousHiddenOutputGradients), (numHidden + 1) * numOutput * sizeof(float));
	hipMalloc((void**) &(propagation->d_inputHiddenLearningRates), (numInput + 1) * numHidden * sizeof(float));
	hipMalloc((void**) &(propagation->d_hiddenOutputLearningRates), (numHidden + 1) * numOutput * sizeof(float));

	// Computed weights
	propagation->h_inputHiddenWeights = (float *) malloc((numInput + 1) * numHidden * sizeof(float));
	propagation->h_hiddenOutputWeights = (float *) malloc((numHidden + 1) * numOutput * sizeof(float));

	// Initialization
	memcpy(propagation->h_inputHiddenWeights, h_inputHiddenWeights, (numInput + 1) * numHidden * sizeof(float));
	memcpy(propagation->h_hiddenOutputWeights, h_hiddenOutputWeights, (numHidden + 1) * numOutput * sizeof(float));

	hipMemcpy(propagation->d_inputsBatch, h_inputData, numSamples * numInput * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(propagation->d_targetOutputsBatch, h_outputData, numSamples * numOutput * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);

	hipMemset(propagation->d_previousInputHiddenWeightDeltas, 0, (numInput + 1) * numHidden * sizeof(float));
	hipMemset(propagation->d_previousHiddenOutputWeightDeltas, 0, (numHidden + 1) * numOutput * sizeof(float));
	hipMemset(propagation->d_previousInputHiddenGradients, 0, (numInput + 1) * numHidden * sizeof(float));
	hipMemset(propagation->d_previousHiddenOutputGradients, 0, (numHidden + 1) * numOutput * sizeof(float));

	randomizeLearningRates(propagation);
	
	return propagation;
}

void destroyErrorPropagation(CudaErrorPropagation *propagation)
{
	if (!propagation)
		return;
	// Network and data
	hipFree(propagation->d_inputsBatch);
	hipFree(propagation->d_inputHiddenWeights);
	hipFree(propagation->d_hiddenOutputsBatch);
	hipFree(propagation->d_hiddenOutputWeights);
	hipFree(propagation->d_outputsBatch);

	// Propagation
	hipFree(propagation->d_targetOutputsBatch);
	hipFree(propagation->d_outputDeltasBatch);
	hipFree(propagation->d_hiddenOutputGradients);
	hipFree(propagation->d_hiddenDeltasBatch);
	hipFree(propagation->d_inputHiddenGradients);
	hipFree(propagation->d_errorsOutputsBatch);
	hipFree(propagation->d_errorsBatch);
	hipFree(propagation->d_error);

	hipFree(propagation->d_hiddenOutputGradientsBatch);
	hipFree(propagation->d_inputHiddenGradientsBatch);

	// BackPropagation
	hipFree(propagation->d_previousInputHiddenWeightDeltas);
	hipFree(propagation->d_previousHiddenOutputWeightDeltas);

	// ResilientPropagation
	hipFree(propagation->d_previousInputHiddenGradients);
	hipFree(propagation->d_previousHiddenOutputGradients);
	hipFree(propagation->d_inputHiddenLearningRates);
	hipFree(propagation->d_hiddenOutputLearningRates);

	// Computed weights
	free(propagation->h_inputHiddenWeights);
	free(propagation->h_hiddenOutputWeights);

	free(propagation);
}

const float* getInputHiddenWeights(CudaErrorPropagation *propagation)
{
	hipMemcpy(propagation->h_inputHiddenWeights, propagation->d_inputHiddenWeights,
		(propagation->numInput + 1) * propagation->numHidden * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	return propagation->h_inputHiddenWeights;
}

const float* getHiddenOutputWeights(CudaErrorPropagation *propagation)
{
	hipMemcpy(propagation->h_hiddenOutputWeights, propagation->d_hiddenOutputWeights,
		(propagation->numHidden + 1) * propagation->numOutput * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	return propagation->h_hiddenOutputWeights;
}

void computeOutputBatch(CudaErrorPropagation *propagation)
{
	dim3 blockDim = getBlockDim2D();

	dim3 gridDim1 = getGridDim2D(propagation->numHidden, blockDim.x, propagation->numSamples, blockDim.y);
	computeLayerOutputBatchKernel<<<gridDim1, blockDim>>>(propagation->d_inputsBatch, propagation->d_inputHiddenWeights,
		propagation->d_hiddenOutputsBatch, propagation->numInput, propagation->numHidden, propagation->numSamples);

	dim3 gridDim2 = getGridDim2D(propagation->numOutput, blockDim.x, propagation->numSamples, blockDim.y);
	computeLayerOutputBatchKernel<<<gridDim2, blockDim>>>(propagation->d_hiddenOutputsBatch, propagation->d_hiddenOutputWeights,
		propagation->d_outputsBatch, propagation->numHidden, propagation->numOutput, propagation->numSamples);
}

void computeGradients(CudaErrorPropagation *propagation)
{
	dim3 blockDim = getBlockDim2D();

	dim3 gridDim1 = getGridDim2D(propagation->numOutput, blockDim.x, propagation->numSamples, blockDim.y);
	computeHOGradsBatchKernel<<<gridDim1, blockDim>>>(propagation->d_hiddenOutputGradientsBatch, propagation->d_errorsOutputsBatch,
		propagation->d_outputDeltasBatch, propagation->d_hiddenOutputsBatch, propagation->d_outputsBatch,
		propagation->d_targetOutputsBatch, propagation->numHidden, propagation->numOutput, propagation->numSamples);

	dim3 gridDim2 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numSamples, blockDim.y);
	computeIHGradsBatchKernel<<<gridDim2, blockDim>>>(propagation->d_inputHiddenGradientsBatch, propagation->d_errorsOutputsBatch,
		propagation->d_errorsBatch, propagation->d_hiddenOutputWeights, propagation->d_outputDeltasBatch,
		propagation->d_hiddenDeltasBatch, propagation->d_hiddenOutputsBatch, propagation->d_inputsBatch,
		propagation->numInput, propagation->numHidden, propagation->numOutput, propagation->numSamples);

	dim3 gridDim3 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numHidden, blockDim.y);
	computeLayerGradsKernel<<<gridDim3, blockDim>>>(propagation->d_inputHiddenGradients, propagation->d_inputHiddenGradientsBatch,
		propagation->d_error, propagation->d_errorsBatch, propagation->d_inputHiddenWeights,
		propagation->numInput, propagation->numHidden, propagation->numSamples, true);

	dim3 gridDim4 = getGridDim2D(propagation->numHidden + 1 /* bias */, blockDim.x, propagation->numOutput, blockDim.y);
	computeLayerGradsKernel<<<gridDim4, blockDim>>>(propagation->d_hiddenOutputGradients, propagation->d_hiddenOutputGradientsBatch,
		propagation->d_error, propagation->d_errorsBatch, propagation->d_hiddenOutputWeights,
		propagation->numHidden, propagation->numOutput, propagation->numSamples, false);
}

void updateWeights(CudaErrorPropagation *propagation, float learningRate, float momentum)
{
	dim3 blockDim = getBlockDim2D();
	
	dim3 gridDim1 = getGridDim2D(propagation->numInput + 1 /* bias */, blockDim.x, propagation->numHidden, blockDim.y);
	updateLayerWeightsBackPropKernel<<<gridDim1, blockDim>>>(propagation->d_inputHiddenGradients, propagation->d_inputHiddenWeights,
		propagation->d_previousInputHiddenWeightDeltas, learningRate, momentum, propagation->numInput, propagation->numHidden);

	dim3 gridDim2 = getGridDim2D(propagation->numHidden + 1 /* bias */, blockDim.x, propagation->numOutput, blockDim.y);
	updateLayerWeightsBackPropKernel<<<gridDim2, blockDim>>>(propagation->d_hiddenOutputGradients, propagation->d_hiddenOutputWeights,
		propagation->d_previousHiddenOutputWeightDeltas, learningRate, momentum, propagation->numHidden, propagation->numOutput);
}

float performBackPropEpoch(CudaErrorPropagation *propagation, float learningRate, float momentum)
{
	computeOutputBatch(propagation);
	computeGradients(propagation);
	updateWeights(propagation, learningRate, momentum);

	float h_error = 100.0f;
	hipError_t status = hipMemcpy(&h_error, propagation->d_error, sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

	//return h_error * 0.5f;
	return sqrtf(h_error / propagation->numSamples);
}